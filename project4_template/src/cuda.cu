#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"

int block_size = 512;     // cuda thread block size
__device__ int size_gpu;  // problem size
int size_cpu;
int n_iter;

__global__ void initialize(float* data) {
    // TODO: intialize the temperature distribution (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size_gpu * size_gpu) {
        data[i] = wall_temp;
    }
}

__global__ void generate_fire_area(bool* fire_area) {
    // TODO: generate the fire area (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = idx / size_gpu;
    int j = idx % size_gpu;

    fire_area[i * size_gpu + j] = 0;
    int a = 0, b = 0, r2 = 0;

    float fire1_r2 = fire_size * fire_size;
    a = i - size_gpu / 2;
    b = j - size_gpu / 2;
    r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
    if (r2 < fire1_r2)
        fire_area[i * size_gpu + j] = 1;

    float fire2_r2 = (fire_size / 2) * (fire_size / 2);
    a = i - 1 * size_gpu / 3;
    b = j - 1 * size_gpu / 3;
    r2 = a * a + b * b;
    if (r2 < fire2_r2)
        fire_area[i * size_gpu + j] = 1;
}

__global__ void update(float* data, float* new_data) {
    // TODO: update temperature for each point  (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size_gpu * size_gpu) {
        if (!(i % size_gpu == 0 || i % size_gpu == size_gpu - 1 ||
              i < size_gpu || i > size_gpu * (size_gpu - 1))) {
            float up = data[i - size_gpu];
            float down = data[i + size_gpu];
            float left = data[i - 1];
            float right = data[i + 1];
            float new_val = (up + down + left + right) / 4;
            new_data[i] = new_val;
        }
    }
}

__global__ void maintain_wall(float* data) {
    // TODO: maintain the temperature of the wall (sequential is enough)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size_gpu * size_gpu) {
        if (i % size_gpu == 0 || i % size_gpu == size_gpu - 1 || i < size_gpu ||
            i > size_gpu * (size_gpu - 1))
            data[i] = wall_temp;
    }
}

__global__ void maintain_fire(float* data, bool* fire_area) {
    // TODO: maintain the temperature of the fire (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size_gpu * size_gpu) {
        if (fire_area[i])
            data[i] = fire_temp;
    }
}

#ifdef GUI
__global__ void data2pixels(float* data, GLubyte* pixels) {
    // TODO: convert rawdata (large, size^2) to pixels (small, resolution^2) for
    // faster rendering speed (in parallelized way)
    float factor_data_pixel = (float)size_gpu / resolution;
    float factor_temp_color = (float)255 / fire_temp;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int x = idx / resolution;
    int y = idx % resolution;
    int idx_pixel = idx * 3;
    int x_raw = x * factor_data_pixel;
    int y_raw = y * factor_data_pixel;
    int idx_raw = y_raw * size_gpu + x_raw;
    float temp = data[idx_raw];
    int color = ((int)temp / 5 * 5) * factor_temp_color;
    pixels[idx_pixel] = color;
    pixels[idx_pixel + 1] = 255 - color;
    pixels[idx_pixel + 2] = 255 - color;
}

void plot(GLubyte* pixels) {
// visualize temprature distribution
#ifdef GUI
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(resolution, resolution, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glutSwapBuffers();
#endif
}
#endif

__global__ void warmup() {}

void master() {
    float* data_odd;
    float* data_even;
    bool* fire_area;

    clock_t clock_start;
    clock_t clock_end;

    hipMalloc(&data_odd, size_cpu * size_cpu * sizeof(float));
    hipMalloc(&data_even, size_cpu * size_cpu * sizeof(float));
    hipMalloc(&fire_area, size_cpu * size_cpu * sizeof(bool));

#ifdef GUI
    GLubyte* pixels;
    GLubyte* host_pixels;
    host_pixels = new GLubyte[resolution * resolution * 3];
    hipMalloc(&pixels, resolution * resolution * 3 * sizeof(GLubyte));
#endif

    int n_block_size = size_cpu * size_cpu / block_size + 1;
    int n_block_resolution = resolution * resolution / block_size + 1;

    initialize<<<n_block_size, block_size>>>(data_odd);
    generate_fire_area<<<n_block_size, block_size>>>(fire_area);

    int count = 1;
    double total_time = 0;

    while (true) {
        // std::chrono::high_resolution_clock::time_point t1 =
        // std::chrono::high_resolution_clock::now();
        clock_start = clock();
        // TODO: modify the following lines to fit your need.
        if (count % 2 == 1) {
            update<<<n_block_size, block_size>>>(data_odd, data_even);
            maintain_fire<<<n_block_size, block_size>>>(data_even, fire_area);
            maintain_wall<<<n_block_size, block_size>>>(data_even);
        } else {
            update<<<n_block_size, block_size>>>(data_even, data_odd);
            maintain_fire<<<n_block_size, block_size>>>(data_odd, fire_area);
            maintain_wall<<<n_block_size, block_size>>>(data_odd);
        }
        clock_end = clock();
        // std::chrono::high_resolution_clock::time_point t2 =
        // std::chrono::high_resolution_clock::now();
        double this_time =
            double(clock_end - clock_start) / double(CLOCKS_PER_SEC);
        // std::chrono::duration<double>(t2 - t1).count();
        total_time += this_time;
        printf("Iteration %d, elapsed time: %.6f\n", count, this_time);
        count++;

#ifdef GUI
        if (count % 2 == 1) {
            data2pixels<<<n_block_resolution, block_size>>>(data_even, pixels);
        } else {
            data2pixels<<<n_block_resolution, block_size>>>(data_odd, pixels);
        }
        hipMemcpy(host_pixels, pixels,
                   resolution * resolution * 3 * sizeof(GLubyte),
                   hipMemcpyDeviceToHost);
        plot(host_pixels);
#endif
        if (count >= n_iter)
            break;
    }

    printf(
        "Converge after %d iterations, elapsed time: %.6f, average computation "
        "time: %.6f\n",
        count - 1, total_time, (double)total_time / (count - 1));

    hipFree(data_odd);
    hipFree(data_even);
    hipFree(fire_area);

#ifdef GUI
    hipFree(pixels);
    delete[] host_pixels;
#endif
}

int main(int argc, char* argv[]) {
    size_cpu = atoi(argv[1]);
    block_size = atoi(argv[2]);
    n_iter = atoi(argv[3]);
    hipMemcpyToSymbol(HIP_SYMBOL(size_gpu), &size_cpu, sizeof(int));

#ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(resolution, resolution);
    glutCreateWindow("Heat Distribution Simulation Sequential Implementation");
    gluOrtho2D(0, resolution, 0, resolution);
#endif

    master();

    printf("Student ID: 120090453\n");  // replace it with your student id
    printf("Name: Haonan XUE\n");       // replace it with your name
    printf("Assignment 4: Heat Distribution CUDA Implementation\n");

    return 0;
}
